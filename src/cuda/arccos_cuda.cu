/*
Compile as follows:
nvcc -arch=sm_90 -o arccos_cuda arccos_cuda.cu
*/

#include <hip/hip_runtime.h>
#include <iostream>

#define N 128 * 128
#define NUM_STREAMS 3

__global__ void compute_kernel(float* d_data, int size, float value) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) d_data[idx] += value;
}

int main() {
    int size = N;
    size_t bytes = size * sizeof(float);

    float* h_data[NUM_STREAMS], *h_result[NUM_STREAMS];
    float* d_data[NUM_STREAMS];
    hipStream_t streams[NUM_STREAMS];

    // Allocate host and device memory, create streams
    for (int i = 0; i < NUM_STREAMS; ++i) {
        hipError_t err = hipHostAlloc(&h_data[i], bytes, hipHostMallocDefault);
        if (err != hipSuccess) {
            std::cerr << "hipHostAlloc failed for h_data[" << i << "]: " << hipGetErrorString(err) << std::endl;
            return 1;
        }
        err = hipHostAlloc(&h_result[i], bytes, hipHostMallocDefault);
        if (err != hipSuccess) {
            std::cerr << "hipHostAlloc failed for h_data[" << i << "]: " << hipGetErrorString(err) << std::endl;
            return 1;
        }
        hipMalloc(&d_data[i], bytes);
        hipStreamCreate(&streams[i]);

        for (int j = 0; j < size; ++j) h_data[i][j] = static_cast<float>(j);
    }

    int threads = 256;
    int blocks = (size + threads - 1) / threads;

    // Launch operations in streams
    for (int i = 0; i < NUM_STREAMS; ++i) {
        hipMemcpyAsync(d_data[i], h_data[i], bytes, hipMemcpyHostToDevice, streams[i]); // HDx
        compute_kernel<<<blocks, threads, 0, streams[i]>>>(d_data[i], size, 1.0f);         // Kx
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "Kernel launch failed in stream " << i << ": " << hipGetErrorString(err) << std::endl;
            return 1;
        }
        hipMemcpyAsync(h_result[i], d_data[i], bytes, hipMemcpyDeviceToHost, streams[i]); // DHx
    }

    // Wait for all streams to finish
    hipDeviceSynchronize();

    // Verify result
    for (int i = 0; i < NUM_STREAMS; ++i) {
        bool correct = true;
        for (int j = 0; j < size; ++j) {
            if (h_result[i][j] != h_data[i][j] + 1.0f) {
                correct = false;
                std::cout << "Mismatch at index " << j << " in stream " << i << ": "
                          << h_result[i][j] << " != " << h_data[i][j] + 1.0f << std::endl;
                break;
            }
        }
        std::cout << "Stream " << i << ": " << (correct ? "Success" : "Failed") << std::endl;
    }

    // Cleanup
    for (int i = 0; i < NUM_STREAMS; ++i) {
        hipHostFree(h_data[i]);
        hipHostFree(h_result[i]);
        hipFree(d_data[i]);
        hipStreamDestroy(streams[i]);
    }

    return 0;
}
